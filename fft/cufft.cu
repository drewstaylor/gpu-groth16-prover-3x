#include "hip/hip_runtime.h"
#include <math.h>
#include <vector>
#include <hipfft/hipfft.h>

static constexpr size_t threads_per_block = 512;

#define NRANK_2D 2
// XXX TODO: Use hipfftPlanMany() in place of hipfftPlan2d() to allow for parellel iFFTs
// https://docs.nvidia.com/cuda/cufft/index.html#cufft-code-examples
template <typename B>
__global__ void
domain_iFFT_single_batch(var *domain, int *ax_Len, int *ay_Len) 
{
    // FFT init types
    hipfftHandle plan;
    hipfftComplex *idata, *odata;
    hipfftResult result;
    int NX = *ax_Len;
    int NY = *ay_Len;
    //int n[NRANK_2D] = {NX, NY};
    
    // GPU allocation and copy domain from CPU into idata / odata
    int input_mem_size = sizeof(hipfftComplex) * NX * NY;
    size_t host_orig_pitch = NX * sizeof(hipfftComplex);
    size_t pitch;

    hipMallocPitch(
        &domain, 
        &pitch, 
        NX * sizeof(hipfftComplex),  // XXX: sizeof(hipfftComplex) may need a custom typedef?
        NY
    );

    /*
    hipMemcpy2D(
        void* dst,                  // Destination memory address
        size_t dpitch,              // Pitch of destination memory
        const void* src,            // Source memory address
        size_t spitch,              // Pitch of source memory
        size_t width,               // Width of matrix transfer (columns in bytes)
        size_t height,              // Height of matrix transfer (rows)
        enum hipMemcpyKind kind    // Type of transfer
    );
    */

    hipMemcpy2D(&idata, pitch, domain, host_orig_pitch, NX* sizeof(hipfftComplex), NY, hipMemcpyHostToDevice);
    hipMemcpy2D(&odata, pitch, domain, host_orig_pitch, NX* sizeof(hipfftComplex), NY, hipMemcpyHostToDevice);
    hipMalloc((void **)&idata, input_mem_size);
    hipMalloc((void **)&odata, input_mem_size);

    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed to allocate\n");
        return;
    }

    // FFT plan creation
    if (hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C) != HIPFFT_SUCCESS) {
        fprintf(stderr, "Cuda error: Plan creation failed");
        return;
    }

    // FFT execution
    result = hipfftExecC2C(plan, idata, odata, HIPFFT_BACKWARD);
    if (result != HIPFFT_SUCCESS) {
        fprintf(stderr, "Cuda error: hipfftExecC2C failed"); // Transformers: "More than meets the eye"
        return;
    }

    // Copy device result to host
    hipMemcpy2D(
        domain, 
        host_orig_pitch, 
        odata, 
        pitch, 
        NX* sizeof(hipfftComplex), 
        NY, 
        hipMemcpyDeviceToHost
    );

    // Clean up
    hipfftDestroy(plan);
    hipFree(idata);
    hipFree(odata);
}

/*
template <typename B>
__global__ void
domain_cosetFFT(var *domain, const var *a)
{
    // XXX TODO: write / convert multiplicative_generator
    //domain->data->cosetFFT(*a->data, Fr<mnt4753_pp>::multiplicative_generator);
}

template <typename B>
__global__ void
domain_icosetFFT(var *domain, const var *a)
{
    // XXX TODO: write / convert multiplicative_generator
    //domain->data->icosetFFT(*a->data, Fr<mnt4753_pp>::multiplicative_generator);
}
*/
