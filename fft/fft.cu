#include "hip/hip_runtime.h"
#include <math.h>
#include <vector>
#include <hipfft/hipfft.h>

/*
B::domain_iFFT(domain, ca);
B::domain_iFFT(domain, cb);

B::domain_cosetFFT(domain, ca);
B::domain_cosetFFT(domain, cb);

B::domain_iFFT(domain, cc);
B::domain_cosetFFT(domain, cc);

B::domain_icosetFFT(domain, H_tmp);



void mnt4753_libsnark::domain_iFFT(mnt4753_libsnark::evaluation_domain *domain,
                                   mnt4753_libsnark::vector_Fr *a) {
  T::CudaVector<Fr<mnt4753_pp>> &data = *a->data;
  domain->data->iFFT(data);
}
void mnt4753_libsnark::domain_cosetFFT(
    mnt4753_libsnark::evaluation_domain *domain,
    mnt4753_libsnark::vector_Fr *a) {
  domain->data->cosetFFT(*a->data, Fr<mnt4753_pp>::multiplicative_generator);
}
void mnt4753_libsnark::domain_icosetFFT(
    mnt4753_libsnark::evaluation_domain *domain,
    mnt4753_libsnark::vector_Fr *a) {
  domain->data->icosetFFT(*a->data, Fr<mnt4753_pp>::multiplicative_generator);
}

*/

// B::domain_iFFT(a, b);
// B::domain_cosetFFT(a, ca);
// B::domain_icosetFFT(a, b);

template<typename B>
class CudaVector {
private:
    T* m_begin;
    T* m_end;

    size_t capacity;
    size_t length;
    
    __device__ void expand() {
        capacity *= 2;
        size_t tempLength = (m_end - m_begin);
        T* tempBegin = new T[capacity];

        memcpy(tempBegin, m_begin, tempLength * sizeof(T));
        delete[] m_begin;
        m_begin = tempBegin;
        m_end = m_begin + tempLength;
        length = static_cast<size_t>(m_end - m_begin);
    }
public:
    __device__  explicit CudaVector() : length(0), capacity(16) {
        m_begin = new T[capacity];
        m_end = m_begin;
    }

    __device__ T& operator[] (unsigned int index) {
        return *(m_begin + index);
    }

    __device__ T* begin() {
        return m_begin;
    }
    
    __device__ T* end() {
        return m_end;
    }

    __device__ ~CudaVector() {
        delete[] m_begin;
        m_begin = nullptr;
    }

    __device__ void add(T t) {

        if ((m_end - m_begin) >= capacity) {
            expand();
        }

        new (m_end) T(t);
        m_end++;
        length++;
    }

    __device__ T pop() {
        T endElement = (*m_end);
        delete m_end;
        m_end--;
        return endElement;
    }

    __device__ size_t getSize() {
        return length;
    }
};

//here
//static constexpr size_t threads_per_block = 1024;
static constexpr size_t threads_per_block = 512;

template <typename B>
__global__ void
domain_iFFT(var *domain, const var *a)
{
    //hipStreamCreateWithFlags(&strm, hipStreamNonBlocking);
    B::CudaVector &data = *a->begin;
    *domain->data->iFFT(&data);
}

/*
template <typename B>
__global__ void
domain_cosetFFT(var *domain, const var *a)
{
    // XXX TODO: write / convert multiplicative_generator
    //domain->data->cosetFFT(*a->data, Fr<mnt4753_pp>::multiplicative_generator);
}

template <typename B>
__global__ void
domain_icosetFFT(var *domain, const var *a)
{
    // XXX TODO: write / convert multiplicative_generator
    //domain->data->icosetFFT(*a->data, Fr<mnt4753_pp>::multiplicative_generator);
}
*/