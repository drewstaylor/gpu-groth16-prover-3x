#include "hip/hip_runtime.h"
#include <math.h>
#include <vector>
#include <hipfft/hipfft.h>

static constexpr size_t threads_per_block = 512;

#define NRANK_2D 2
// XXX TODO: Use hipfftPlanMany() in place of hipfftPlan2d() to allow for parellel iFFTs
// https://docs.nvidia.com/cuda/cufft/index.html#cufft-code-examples
template <typename B>
__global__ void
domain_iFFT_single_batch(var *domain, int *ax_Len, int *ay_Len, const var *aX, const var *aY) 
{
    // FFT init types
    hipfftHandle plan;
    hipfftComplex *data; // XXX: Or may need to create *data_in, *data_out TBD
    hipfftResult result;
    int NX = *ax_Len;
    int NY = *ay_Len;
    int n[NRANK_2D] = {NX, NY};
    
    // GPU allocation and copy domain from CPU into data
    int input_mem_size = sizeof(hipfftComplex) * NX * NY;
    size_t host_orig_pitch = NX * sizeof(hipfftComplex);
    size_t pitch;

    hipMallocPitch(
        &domain, 
        &pitch, 
        NX * sizeof(hipfftComplex),  // XXX: sizeof(hipfftComplex) may need a custom typedef?
        NY
    );

    /*
    hipMemcpy2D(
        void* dst,                  // Destination memory address
        size_t dpitch,              // Pitch of destination memory
        const void* src,            // Source memory address
        size_t spitch,              // Pitch of source memory
        size_t width,               // Width of matrix transfer (columns in bytes)
        size_t height,              // Height of matrix transfer (rows)
        enum hipMemcpyKind kind    // Type of transfer
    );
    */

    hipMemcpy2D(data, pitch, domain, host_orig_pitch, NX* sizeof(hipfftComplex), NY, hipMemcpyHostToDevice);
    hipMalloc((void **)&data, input_mem_size);

    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed to allocate\n");
        return;
    }

    // FFT plan creation
    if (hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C) != HIPFFT_SUCCESS) {
        fprintf(stderr, "Cuda error: Plan creation failed");
        return;
    }

    // FFT execution
    result = hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);
    if (result != HIPFFT_SUCCESS) {
        fprintf(stderr, "Cuda error: hipfftExecC2C failed"); // Transformers: "More than meets the eye"
        return;
    }

    // Copy device result to host
    hipMemcpy2D(
        domain, 
        host_orig_pitch, 
        data, 
        pitch, 
        NX* sizeof(hipfftComplex), 
        NY, 
        hipMemcpyDeviceToHost
    );

    // Clean up
    hipfftDestroy(plan);
    hipFree(data);
}

/*
template <typename B>
__global__ void
domain_cosetFFT(var *domain, const var *a)
{
    // XXX TODO: write / convert multiplicative_generator
    //domain->data->cosetFFT(*a->data, Fr<mnt4753_pp>::multiplicative_generator);
}

template <typename B>
__global__ void
domain_icosetFFT(var *domain, const var *a)
{
    // XXX TODO: write / convert multiplicative_generator
    //domain->data->icosetFFT(*a->data, Fr<mnt4753_pp>::multiplicative_generator);
}
*/